/*
initLibs.cu
Initialize libraries to use.

Nicolas Sawaya
2013
*/



bool initCula() {
    culaStatus culaStat;
    
    culaStat = culaInitialize();
    if(culaStat!=culaNoError) {
        cout << "CULA failed to initialize. " << endl;;
        cout << "culaGetStatusString(culaStat) = " << culaGetStatusString(culaStat);
        cout << endl;
        cout << "culaGetErrorInfo() = " << culaGetErrorInfo();
        //Apparently above returns error integer equivalent to LAPACK error
        cout << ". Aborting." << endl;
        return false;
    }
    return true;
    
}



bool initCusparse(hipsparseHandle_t *cusparseHandle) {
    hipsparseStatus_t csStatus;
    csStatus = hipsparseCreate(cusparseHandle);
    if (csStatus != HIPSPARSE_STATUS_SUCCESS) {
        cout << "Error from hipsparseCreate(). csStatus = ";
        cout << hipsparseGetErrorString(csStatus) << endl;
        return false;
    }
    //Set pointer mode to device
    hipsparseSetPointerMode(*cusparseHandle, HIPSPARSE_POINTER_MODE_DEVICE);
    return true;
}


bool initCublas(hipblasHandle_t *cublasHandle) {
    hipblasStatus_t cbStatus;
    //cublasHandle = 0;
    cbStatus = hipblasCreate(cublasHandle);
    if (cbStatus != HIPBLAS_STATUS_SUCCESS) {
        cout << "Error from hipblasCreate(). cbStatus = ";
        cout << cublasGetErrorString(cbStatus) << endl;
        return false;
    }
    //Set pointer mode to device
    cbStatus = hipblasSetPointerMode(*cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);

    return true;
}


bool initCufft(hipfftHandle *cufftPlan, int arrlength) {

    //hipfftPlan1d(plan, nx, cuffttype, batch)
    if( hipfftPlan1d(cufftPlan, arrlength, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS ) {
        cout << "ERROR in initCufft with hipfftPlan1d(). Aborting." << endl;
        return false;
    }

    return true;

}




















