#include "hip/hip_runtime.h"
/*
taylorKrylovClass.cu
Class for Taylor-Krylov method.

Nicolas Sawaya
September 2013
*/



template <typename tMat, typename tState, typename tReal>
class taylorKrylovClass {

    public:
        taylorKrylovClass();
        taylorKrylovClass(
            systemClass<tMat,tState,tReal> *sysObjPtr,
            matCsrClass<tMat> *d_AmatCsr,
            string matBFileName,
            int p,
            int m_forBmat,
            tMat h_expCoeff,
            tMat *d_expCoeff
        );
        //void set_m(int m);


        //Object pointer (avoid using this altogether)
        systemClass<tMat,tState,tReal> *sysObjPtr;

        //Objects (not pointers)
        lanczosClass<tMat,tState,tReal> *lanObjBPtr;    //for B-matrix
        padeClass<tMat,tState,tReal> *padeObjBPtr;      //for B-matrix

        //Pointer to Hamiltonian


        bool setupProblem(); //don't know if necessary, but probably is

        bool propagateWithTaylorKrylov(tMat* ptrInVec, tMat* ptrOutVec);
        //bool doTaylorPart();
        //bool doKrylovPart();

        tReal getTaylorKrylovError(tReal vecNorm);

        bool sparseMatDenseVecMult(matCsrClass<tMat> d_csrMat, tMat* inVec, tMat* resultVec);
        bool do_Apowk_times_input(int k);


        void cleanup();


        //Size of system is N, size of decomposed matrix is m
        int N;
        int m_forBmat; //Used only on Krylov part
        int p; //This is number of 
        //int lenMatT; //This will be m+2 because of error calculation routine


        //Matrix pointers
        matCsrClass<tMat> *d_AmatCsr;

        //Matrices in memory
        matCooClass<tMat> h_BmatCoo;
        matCooClass<tMat> d_BmatCoo;
        matCsrClass<tMat> h_BmatCsr;
        matCsrClass<tMat> d_BmatCsr; //<-- this is the one we're prepping for

        //Vectors for calculating Taylor series
        //Switching back and forth between the two 'taylor components'
        thrust::device_vector<tMat> d_taylorCoeffs;
        thrust::device_vector<tMat> d_taylorComponent1;
        thrust::device_vector<tMat> d_taylorComponent2;
        tMat* d_taylorCompRawPtr1;
        tMat* d_taylorCompRawPtr2;

        //Vectors holding both parts of calculation
        thrust::device_vector<tMat> d_taylorPartVec;
        thrust::device_vector<tMat> d_krylovPartVec;
        //PROBABLY MERGE THESE

        //Pointers to the in and out vectors
        tMat* vecInPtr;
        tMat* vecOutPtr;

        //Pointers to zero-val and one-val
        tMat* d_zeroComplex;
        tMat* d_oneComplex;

        //This is the matrix coefficient, that I'm calling 'tau'
        tMat h_expCoeff;
        tMat* d_expCoeff;


        //h_m+1,m in Expokit paper, equals beta[m]
        //tMat h_mp1_m;

        // ||A*v_{m+1}||_2, used in error analysis
        //tReal norm2_A_v_mp1;



        //Constants for real parts
        //typeReal *d_vecNorm, *d_invVecNorm;

        //Which state vector (in other words, which time step)
        //int vecInNum;


        //CONST MEMORY???
        //Constants used in matrix operations
        //tMat  h_zero,  h_one,  h_two,  h_negOne, h_imag;
        //tMat *d_zero, *d_one, *d_two, *d_negOne, *d_imag;
        //tMat h_hbar_inv_cm_fs, h_hbar_J_s, h_lightspeed, h_planck_J_s;
        //tMat h_neg_inv_hbar_imag, *d_neg_inv_hbar_imag;




};



//Default Constructor
template <typename tMat, typename tState, typename tReal>
taylorKrylovClass<tMat,tState,tReal>::
taylorKrylovClass() {

}


//Constructor
template <typename tMat, typename tState, typename tReal>
taylorKrylovClass<tMat,tState,tReal>::
taylorKrylovClass(
    systemClass<tMat,tState,tReal> *sysObjPtr,
    matCsrClass<tMat> *d_AmatCsr,
    string matBFileName,
    int p,
    int m_forBmat,
    tMat h_expCoeff,
    tMat *d_expCoeff
    ) {


    //Avoid using this altogether
    this->sysObjPtr = sysObjPtr;

    //Assign expCoeff
    this->h_expCoeff = h_expCoeff;
    this->d_expCoeff = d_expCoeff;

    //Point to the main matrix ("A")
    this->d_AmatCsr = d_AmatCsr;

    //Read in matBFilename
    if( ! this->h_BmatCoo.readInFile(matBFileName, false) ) {
        cout << "ERROR in taylorKrylovClass constructor, reading in file." << endl;
    }
    
    //Set up the B-matrix. Prob want to switch matrix type later.
    this->d_BmatCoo.setCusparseHandle(& sysObjPtr->csHandle);
    this->h_BmatCoo.setCusparseHandle(& sysObjPtr->csHandle);
    this->d_BmatCsr.setCusparseHandle(& sysObjPtr->csHandle);
    this->h_BmatCsr.setCusparseHandle(& sysObjPtr->csHandle);

    //Set up Hamiltonian on system
    this->d_BmatCoo.createOnDevice(this->h_BmatCoo);
    //Copy to csr on device
    this->d_BmatCsr.pointToCooAndConvert(this->d_BmatCoo);
    //Copy csr to host. Don't do this unless testing.
    //if( ! this->h_BmatCsr.createFromDeviceCsrMat(this->d_BmatCsr) ) return false;


    //Copy system parameters over
    this->N = sysObjPtr->N;
    this->p = p;
    this->m_forBmat = m_forBmat;


    //Initialize internal lanczos and pade objects
    this->lanObjBPtr = new lanczosClass<tMat,tState,tReal>(
            sysObjPtr, this->m_forBmat, &(this->d_BmatCsr), this->p );     //for B-matrix

    this->padeObjBPtr =  new padeClass<tMat,tState,tReal>(sysObjPtr, this->lanObjBPtr, this->d_expCoeff);      //for B-matrix

    //Initialize vectors
    d_taylorCoeffs.resize(this->p);
    d_taylorComponent1.resize(this->N);
    d_taylorComponent2.resize(this->N);
    d_taylorPartVec.resize(this->N);
    d_krylovPartVec.resize(this->N);

    //Have these pointers for convenience later on
    d_taylorCompRawPtr1 = thrust::raw_pointer_cast(&d_taylorComponent1[0]);
    d_taylorCompRawPtr2 = thrust::raw_pointer_cast(&d_taylorComponent2[0]);

    //Populate Taylor coefficients.
    //Have 1,tau,tau^2/2,...,tau^(p-1)/(p-1)!,[zero] so that you can have 0 and 1 in there
    thrust::host_vector<tMat> h_taylorCoeffs;
    h_taylorCoeffs.resize(this->p+1);
    for(int k=0;k<p;k++) {
        h_taylorCoeffs[k] = powComplex(h_expCoeff,k);
        h_taylorCoeffs[k].x /= helperFactorial(k);
        h_taylorCoeffs[k].y /= helperFactorial(k);
        cout << "h_taylorCoeffs["<<k<<"].{x,y} = " << h_taylorCoeffs[k].x << "," << h_taylorCoeffs[k].y << endl;
    }
    h_taylorCoeffs[p].x = 0.; h_taylorCoeffs[p].y = 0.;

    //Copy constants to device
    d_taylorCoeffs = h_taylorCoeffs;


    //Just point to the values in the taylor series for the zero and one coefficients
    this->d_zeroComplex = thrust::raw_pointer_cast(&this->d_taylorCoeffs[p]);
    this->d_oneComplex = thrust::raw_pointer_cast(&this->d_taylorCoeffs[0]);

}

//Function to call when propagating with Taylor-Krylov
template <typename tMat, typename tState, typename tReal>
bool taylorKrylovClass<tMat,tState,tReal>::
propagateWithTaylorKrylov(tMat* ptrInVec, tMat* ptrOutVec) {

    //Update vector pointers in object
    this->vecInPtr = ptrInVec;
    this->vecOutPtr = ptrOutVec;





    // ** Do Taylor part first (up to A^(p-1)*vecIn) **

    //For first calculation, do A*w
    if(!
    sparseMatDenseVecMult(  //result = Mat*source. Asynchronous.
        *(this->d_AmatCsr),          //mat
        this->vecInPtr,             //source
        d_taylorCompRawPtr1    //result
        )
    ) return false;

    //Copy vector over to the taylorSum vector
    hipblasStatus_t cbStatus;
    cbStatus = hipblasZcopy(
        this->sysObjPtr->cbHandle,
        this->N,     //vector length
        ptrInVec,    //source
        1,  //stride
        thrust::raw_pointer_cast(&d_taylorPartVec[0]),   //result
        1   //stride
    );
    if(cbStatus != HIPBLAS_STATUS_SUCCESS) { 
        cout << "ERROR with hipblasZcopy in propagateWithTaylorKrylov. ";
        cout << "Aborting." << endl;
        return false;
    }

    //Synchronize, because csrmv() is asynchronous
    hipDeviceSynchronize();
    
    //Pointers to switch between
    tMat* ptrTaylorCompResult = this->d_taylorCompRawPtr2;
    tMat* ptrTaylorCompSource = this->d_taylorCompRawPtr1; //Corresponds to A*vec


    //Start looping
    for( int k=2; k <= (this->p); k++ ) {

        //Compute A^k with cusparse (asynchronous)
        if(!
        sparseMatDenseVecMult(  //result = Mat*source. Asynchronous.
            *(this->d_AmatCsr),     //mat
            ptrTaylorCompSource,    //source
            ptrTaylorCompResult     //result
            )
        ) return false;

        //Do taylorSum = taylorSum + coeff*A^(k-1)*w
        hipblasZaxpy(    //y = y + alpha*x
            this->sysObjPtr->cbHandle,  //handle
            this->N,                    //vector length
            thrust::raw_pointer_cast(&this->d_taylorCoeffs[k-1]), //alpha
            ptrTaylorCompSource,     //x. This is A^(k-1).
            1,  //stride
            thrust::raw_pointer_cast(&d_taylorPartVec[0]),    //y. The taylorSum.
            1    //stride
        );

        //Synchronize, because cusparse command was asynchronous
        hipDeviceSynchronize();

        //Update pointer being used
        if(ptrTaylorCompResult == this->d_taylorCompRawPtr1) {
            ptrTaylorCompResult = this->d_taylorCompRawPtr2;
            ptrTaylorCompSource = this->d_taylorCompRawPtr1;
        } else if(ptrTaylorCompResult == this->d_taylorCompRawPtr2) {
            ptrTaylorCompResult = this->d_taylorCompRawPtr1;
            ptrTaylorCompSource = this->d_taylorCompRawPtr2;
        } else {
            cout << "ERROR in propagateWithTaylorKrylov updating pointer!!" << endl;
            return false;
        }

    }


    //Calculate norm, which is used for error calculation



    //Copy back and print taylorpart to test
    // thrust::host_vector<tMat> hostVec;
    // hostVec.resize(d_taylorPartVec.size());
    // hostVec = d_taylorPartVec;
    // cout << "Taylor part: " << endl;
    // for(int elem=0 ; elem<hostVec.size() ; elem++) {
    //     cout << setw(14) << hostVec[elem].x << "  " << hostVec[elem].y;
    //     cout << " i ("<<elem<<")" << endl;
    // }


    //Check value of A^n*vec
    // hipMemcpy(thrust::raw_pointer_cast(&hostVec[0]),ptrTaylorCompSource,
    //     this->N * sizeof(hostVec[0]),
    //     hipMemcpyDeviceToHost);
    // cout << "A^n*vec: " << endl;
    // for(int elem=0 ; elem<hostVec.size() ; elem++) {
    //     cout << setw(14) << hostVec[elem].x << "  " << hostVec[elem].y;
    //     cout << " i ("<<elem<<")" << endl;
    // }





    // ** Do Krylov part **

    //Do lanczos decomposition
    lanObjBPtr->doLanczos(ptrTaylorCompSource, true /* = calcAndStoreNormAndScale */); //Using pointer for A^p*vec

    //Copy back and print norm of A^n*vec, to test
    // tMat normVal = lanObjBPtr->copyBackAndGetNorm();
    // cout << "Norm of A^n*vec = " << normVal.x << "  " << normVal.y << endl;


    //Exponentiate the augmented matrix
    padeObjBPtr->doPade();

    //Copy back and print matExp_tT to test
    // padeObjBPtr->matExp_tT.copyThisDeviceToThisHost();
    // cout << "matExp_tT matrix:" << endl;
    // padeObjBPtr->matExp_tT.printHostMat();

    //Get pointer for last column in matrix, which is equal to tau^p*phi_n()*e1
    tMat* ptrPhi_n = padeObjBPtr->matExp_tT.getDeviceColumnPtr(this->m_forBmat + this->p - 1);
    tMat* ptrQ = lanObjBPtr->matQ.getDeviceColumnPtr(1);

    //Do multiplication and put Krylov vector into place
    //taylorPart = Q_m * lastColumn
    cbStatus = hipblasZgemv( //y = alpha*op(A)*x + beta*y
        sysObjPtr->cbHandle,
        HIPBLAS_OP_N,
        this->N, //rows in mat
        this->m_forBmat, //cols in mat
        this->lanObjBPtr->d_norm, //** 2-norm of original vector **
        ptrQ, //The matrix (first column is just zeros)
        this->N, //leading dimension of matrix (number of elements per column)
        ptrPhi_n, //vector (first column of exp(tT) matrix)
        1, //stride
        lanObjBPtr->d_zero, //beta-multiplier zero.
        ptrOutVec, //result vector
        1 //stride
    );
    if(cbStatus!=HIPBLAS_STATUS_SUCCESS) {
        cout << "ERROR at hipblasZgemv() in taylorKrylovClass::propagateSystem()." << endl;
        return false;
    }


    //Copy back and print krylov part to test
    // hipMemcpy(thrust::raw_pointer_cast(&hostVec[0]),ptrOutVec,
    //     this->N * sizeof(hostVec[0]),
    //     hipMemcpyDeviceToHost);
    // cout << "Krylov part: " << endl;
    // for(int elem=0 ; elem<hostVec.size() ; elem++) {
    //     cout << setw(14) << hostVec[elem].x << "  " << hostVec[elem].y;
    //     cout << " i ("<<elem<<")" << endl;
    // }


    //Add the taylor part (d_taylorPartVec) to the result
    cbStatus = hipblasZaxpy(    //y = y + alpha*x
        this->sysObjPtr->cbHandle,  //handle
        this->N,                    //vector length
        lanObjBPtr->d_one,          //alpha
        thrust::raw_pointer_cast(&d_taylorPartVec[0]),      //x.
        1,  //stride
        ptrOutVec,    //y.
        1    //stride
    );
    if(cbStatus!=HIPBLAS_STATUS_SUCCESS) {
        cout << "ERROR adding taylor and krylov parts in taylorKrylovClass::propagateSystem()." << endl;
        return false;
    }


    return true;


}


//Single matrix multiplication
template <typename tMat, typename tState, typename tReal>
tReal taylorKrylovClass<tMat,tState,tReal>::
getTaylorKrylovError(tReal vecNorm) {

    tReal errEst;

    //Krylov part of error
    errEst = this->padeObjBPtr->getKrylovError();

    //Print to test
    cout << "Krylov part of error for this step: " << errEst << endl;
    cout << "Norm-estimate error for this step: " << abs(1.-vecNorm) << endl;

    //Norm error
    errEst = errEst + abs(1.-vecNorm);

    return errEst;

}


//Single matrix multiplication
template <typename tMat, typename tState, typename tReal>
bool taylorKrylovClass<tMat,tState,tReal>::
sparseMatDenseVecMult(matCsrClass<tMat> d_csrMat, tMat* inVec, tMat* resultVec) {

    //cout << "Beginning lanczosClass.sparseMatDenseVecMult()." << endl;

    //Creating a handle just to see if it will work
    // hipsparseHandle_t Xcshandle=0;
    // hipsparseCreate(&Xcshandle);
    // hipsparseSetPointerMode(Xcshandle, HIPSPARSE_POINTER_MODE_DEVICE);

    hipsparseStatus_t csStatus;
    csStatus = hipsparseZcsrmv( //y = alpha*op(A)*x + beta*y //ASYNCHRONOUS!

        sysObjPtr->csHandle,                    //handle
        //Xcshandle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,   //operation
        d_csrMat.N,                         //Rows in mat A
        d_csrMat.N,                         //Columns in mat A
        d_csrMat.nnz,                       //nnz
        d_oneComplex,                              //constant
        d_csrMat.cuspMatDescr,              //matrix descriptor
        d_csrMat.csrValA,                   //values in csr matrix
        d_csrMat.csrRowPtrA,                   //csrRowPtrA
        d_csrMat.csrColIndA,                   //csrColIndA
        inVec,                              //x-vector
        d_zeroComplex,                             //the beta-value
        resultVec                           //y-vector (result)

    );
    if(csStatus!=HIPSPARSE_STATUS_SUCCESS) {
        cout << "ERROR in sparseMatDenseVecMult(). ";
        cout << "Error was: " << hipsparseGetErrorString(csStatus);
        cout << ". Aborting" << endl;
        cout << endl;
        return false;
    }

    //Synchronize, because csrmv() is asynchronous
    hipDeviceSynchronize();

    return true;

}


// //Single matrix multiplication
// template <typename tMat, typename tState, typename tReal>
// bool taylorKrylovClass<tMat,tState,tReal>::
// sparseMatDenseVecMult(matCsrClass<tMat> d_csrMat, tMat* inVec, tMat* resultVec) {

//     //cout << "Beginning lanczosClass.sparseMatDenseVecMult()." << endl;

//     //Creating a handle just to see if it will work
//     // hipsparseHandle_t Xcshandle=0;
//     // hipsparseCreate(&Xcshandle);
//     // hipsparseSetPointerMode(Xcshandle, HIPSPARSE_POINTER_MODE_DEVICE);

//     hipsparseStatus_t csStatus;
//     csStatus = hipsparseZcsrmv( //y = alpha*op(A)*x + beta*y //ASYNCHRONOUS!

//         sysObjPtr->csHandle,                    //handle
//         //Xcshandle,
//         HIPSPARSE_OPERATION_NON_TRANSPOSE,   //operation
//         d_csrMat.N,                         //Rows in mat A
//         d_csrMat.N,                         //Columns in mat A
//         d_csrMat.nnz,                       //nnz
//         d_one,                              //constant
//         d_csrMat.cuspMatDescr,              //matrix descriptor
//         d_csrMat.csrValA,                   //values in csr matrix
//         d_csrMat.csrRowPtrA,                   //csrRowPtrA
//         d_csrMat.csrColIndA,                   //csrColIndA
//         inVec,                              //x-vector
//         d_zero,                             //the beta-value
//         resultVec                           //y-vector (result)

//     );
//     if(csStatus!=HIPSPARSE_STATUS_SUCCESS) {
//         cout << "ERROR in sparseMatDenseVecMult(). ";
//         cout << "Error was: " << hipsparseGetErrorString(csStatus);
//         cout << ". Aborting" << endl;
//         cout << endl;
//         return false;
//     }

//     //Synchronize, because csrmv() is asynchronous
//     hipDeviceSynchronize();

//     return true;

// }


































