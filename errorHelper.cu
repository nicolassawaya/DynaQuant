/*
errorHelper.cu
Prints out enumerated errors from cusparse and cublas.

Nicolas Sawaya
2013
*/


const char* cublasGetErrorString(hipblasStatus_t status){
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "error not enumerated";
}



const char* hipsparseGetErrorString(hipsparseStatus_t status) {
    switch(status)
    {
        case HIPSPARSE_STATUS_SUCCESS: return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED: return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED: return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE: return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH: return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR: return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED: return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR: return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    }
    return "error not enumerated";

}



const char* cufftGetErrorString(hipfftResult_t status) {
    switch(status)
    {
        case HIPFFT_SUCCESS: return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN: return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED: return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE: return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED: return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED: return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE: return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";
    } 

    
    return "error not enumerated";

}












